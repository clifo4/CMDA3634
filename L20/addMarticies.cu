#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<hip/hip_runtime.h>

//device function
__global__ void kernelAddMatrices1D(int N, double *A, double *B, double *C) {
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int blockSize = blockDim.x;

  int id = thread + blockId*blockSize;

  C[id] = A[id] + B[id];
}

__global__ void kernelAddMatrices2D(int N, double *A, double *B, double *C) {
  int tIdx = threadIdx.x;
  int tIdy = threadIdx.y;
  int bIdx = blockIdx.x;
  int bIdy = blockIdx.y;

  int bSizex = blockDim.x;
  int bSizey = blockDim.y;

  int i= tIdx + bIdx*bSizex;
  int j= tIdy + bIdy*bSizey;
  
  nx = 1024;

  C[i+j*nx] = A[i+j*nx] + B[i+j*nx];

}

__global__ void kernelMatrixTranspose2D_v1(double *A, double *At) {
  
  int tIdx = threadIdx.x;
  int tIdy = threadIdx.y;
  int bIdx = blockIdx.x;
  int bIdy = blockIdx.y;

  int bSizex = blockDim.x;
  int bSizey = blockDim.y;


  int i= tIdx + bIdx*bSizex;
  int j= tIdy + bIdy*bSizey;
  
  nx = 1024;

  At[i+j*nx] = A[j+i*nx];
}


__global__ void kernelMatrixTranspose2D_v2(double *A, double *At) {
  
  int tIdx = threadIdx.x;
  int tIdy = threadIdx.y;
  int bIdx = blockIdx.x;
  int bIdy = blockIdx.y;

  int bSizex = blockDim.x;
  int bSizey = blockDim.y;

  __shared__ double s_A[32][32];

  int i= tIdx + bIdx*bSizex;
  int j= tIdy + bIdy*bSizey;
  
  nx = 1024;
  //fetch a block of A into shared array
  s_A[tIdx][tIdy] = A[i+j*nx]; //read from A and write the block's transpose
  
  __syncthreads(); //barrier the threads on this block
  
  At[i+j*nx] = s_A[tIdy][tIdx];
}

int main(int argc, char** argv){
  int nx = 1024;
  int ny = 1024;

  int N = nx*ny;

  double seed = clock();
  srand48(seed);

  double *h_a, *h_b, *h_c;
  h_a= (double*) malloc(N*sizeof(double));
  h_b= (double*) malloc(N*sizeof(double));
  h_c= (double*) malloc(N*sizeof(double));
  
  for(int n=0; n<N; n++) {
    h_a[n] = drand48();
    h_b[n] = drand48();
  }

  double hostStart = clock();

  for(int j=0; j<ny; j++) {
    for(int i=0; i<nx; i++) {
      h_c[j*ny+i] = h_a[j*ny+i] + h_b[j*ny+i];
    }
  }

  double hostEnd = clock();

  double hostTime = (hostEnd-hostStart)/(double)CLOCKS_PER_SEC;
  size_t inputMem = 2*N*sizeof(double);
  size_t outputMem = 1*N*sizeof(double);
  size_t totalMem_GB = (inputMem + outputMem)/0x20000000;
  printf("The host took %f seconds to computer\n", hostTime)
  
  //Device arrays
  double *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N*sizeof(double));
  hipMalloc(&d_b, N*sizeof(double));
  hipMalloc(&d_c, N*sizeof(double));

  double copyStart = clock();
  hipMemcpy(d_a, //destination
              h_a,  //source
              N*sizeof(double), //size 
              hipMemcpyHostToDevice);  //Command
  hipMemcpy(d_b,h_b,N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_c,h_c,N*sizeof(double), hipMemcpyHostToDevice);
  double copyEnd = clock();
  double copyTime = (copyEnd - copyStart)/(double) CLOCKS_PER_SEC;
  printf("Copying took %f seconds", copyTime);

  
  int nThreads = 32;
  int nBlocks = (N+nThreads-1)/nThreads;
  
  double deviceStart = clock();

  kernellAddVectors <<< nBlocks,nThreads >>>(d_a, d_b, d_c, N);
  
  hipDeviceSynchronize();

  double deviceEnd = clock();
  double deviceTime = (deviceEnd - deviceStart)/(double) CLOCKS_PER_SEC;

  printf("The device took %f seconds", deviceTime);

  //use 2D thread blocks instead
  int Nthreadsx = 32;
  int Nthreadsy = 32;
  int Nthreadsz = 1;
  
  //declare size of block
  //Nthreadsx*Nthreadsy*Nthreadsz <= 1024
  dim3 Nthreads3(Nthreadsx, Nthreadsy, Nthreadsz);
  
  int Nblocksx = (nx+Nthreadsx-1)/Nthreasdsx;
  int Nblocksy = (ny+Nthreadsy-1)/Nthreasdsy;
  int Nblocksz = 1;
  dim3 Nblocks3(Nblocksx, Nblocksy, Nblocksz);
  kernel AddMatrices2D <<< Nblocks, Nthreads3 >>> (N, d_a, d_b, d_c);

  hipDeviceSynchronize();



  kernelMatrixTranspose2d_v1 <<< Nblocks3, Nthreads3 >>>(d_a, d_c);

  hipDeviceSynchronize();





  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);
}
