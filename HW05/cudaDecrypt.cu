#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int cumodprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int cumodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = cumodprod(aExpb, z, p);
    z = cumodprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void cuDecrypt(unsigned int p, unsigned int g, unsigned int h, unsigned int max, unsigned int *x) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bSize = blockDim.x;
  int id = tid + bim*bSize + 1;
  int tSize = gridDim.x*bSize;
  __shared__ unsigned int *re = &0;
  while(id < max-tSize && re[0]==0) {
    if(modExp(g, id, p)) {
      re[0]=id;
      x[0] = id;
    }
    id+=tSize;
  }
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */


  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  
  char* keyFile = "public_key.txt";
  FILE *keys = fopen(keyFile, "r");
  fscanf(keys, "%u %u %u %u", &n, &p, &g, &h);
  fclose(keys);
  unsigned int *m, *a;
  char* enname = "message.txt";
  FILE *enfile = fopen(enname, "r");
  fscanf(enfile, "%u", &Nints);
  m=(unsigned int*)malloc(Nints*sizeof(int));
  a=(unsigned int*)malloc(Nints*sizeof(int));
  for(int i=0; i<Nints; i++) {
    fscanf(enfile, "%u %u", &m[i], &a[i]);
  }
  fclose(enfile);
  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    int max = 1;
    for(int i=0; i<n; i++) {max *=2}
    max = max-1;
    int nThreads = 32;
    int nBlocks = 10;
    unsigned int *d_x;
    hipMemcpy(d_x, &0, 1, hipMemcpyHostToDevice);
    double startTime = clock();
    
    cudecrypt <<< nThreads, nBlocks >>>(p, g, h, max, d_x);
    hipDeviceSynchronize();

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  ElGamalDecrypt(m, a, Nints, p, x);
  int charsperint = n%9;
  int chars = charsperint*Nints;
  char* message = (char*)malloc(chars*sizeof(char)+2);
  convertZToString(m, Nints, message, chars);
  printf("%s\n", message);
  free(message);
  free(m);
  free(a);
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
