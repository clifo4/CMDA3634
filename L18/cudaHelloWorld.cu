#include<stdio.h>
#include <stdlib.h>
#include<hip/hip_runtime.h>

//This is my DEVICE function
//__global__ means that this function is visible to the host
__global__ void kernelHelloWorld() {

  int thread = threadIdx.x;//local thread number in a block
  int block = blockIdx.x;//block number

  printf("Hello World from thread %d of block %d!\n", thread, block);
}

int main(int argc, char** argv) {

  int Nblocks = 10;
  int Nthreads = 3;
  //un the function 'kernelHelloWorld' on the DEVICE
  kernelHelloWorld <<< Nblocks, Nthreads >>> ();
 
 //wait for the DEVICE function to complete before moving on 
  hipDeviceSynchronize();

  return 0;
}
