#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>


__global__ void reduction(float N,float* d_a, float* result) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bSize = blockDim.x;
  int id = tid + bid*bSize;

  __shared__ float s_sum[256];
  if(id<N) {
  s_sum[tid] = a[id];
  } else {
    s_sum[tid]=0;
  }
  __syncthreads();


  if(tid<128) {
    s_sum[tid] += s_sum[tid+128];
  }
  
  __syncthreads();

  if(tid<64) {
    s_sum[tid] += s_sum[tid+64];
  }
  
  __syncthreads();
  if(tid<32) {
    s_sum[tid] += s_sum[tid+32];
  }
  
  __syncthreads();
  if(tid<16) {
    s_sum[tid] += s_sum[tid+16];
  }
  
  __syncthreads();

  if(tid<8) {
    s_sum[tid] += s_sum[tid+8];
  }
  
  __syncthreads();
  if(tid<4) {
    s_sum[tid] += s_sum[tid+4];
  }
  
  __syncthreads();

  if(tid<2) {
    s_sum[tid] += s_sum[tid+2];
  }
  
  __syncthreads();
  if(tid<1) {
    s_sum[tid] += s_sum[tid+1];
    result[bid] = s_sum[tid];
  }
  /*
  if(id<N) {
    *result += d_a[id];
  }*/
}

//perform a reduction of vector length N
int main(int argc, char** argv) {
  int N = atoi(argv[1]);
  double seed = 0;
  srand48(seed);

  float* h_a = (float*) malloc(N*sizeof(float));

  //populate array with randoms
  for(int n=0; n<N; n++) {
    h_a[n] = drand48();
  }
  float h_sum = 0.;
  for(int n=0; n<N; n++) {
    h_sum += h_a[n];
  }

  printf("The host sum was %f \n", h_sum);

  float *d_a, *d_sum;

  int Nnew = (N+256-1)/256;


  hipMalloc(%d_a, N*sizeof(float));
  hipMalloc(%d_sum, Nnew*sizeof(float));

  float *h_nSum = (float*) malloc(Nnew*sizeof(float));
  
  cudamMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
  
  do{
    
    Nnew = (N+256-1)/256;
    dim3 B(256, 1, 1);
    dim3 G(Nnew, 1, 1);

    reduction <<< G, B >>> (N, d_a, d_sum);
    
    cudaMemcp(d_a, d_sum, Nnew*sizeof(float), hipMemcpyDeviceToDevice);

    n = Nnew;
  } while(Nnew>1); 
  hipMemcpy(h_nSum, d_sum, Nnew*sizeof(float), hipMemcpyDeviceToHost);

  printf("The device sum was %f \n", *h_nSum);
  return 0;
}
